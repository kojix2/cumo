#include "hip/hip_runtime.h"
#if defined(__cplusplus)
#if 0
{ /* satisfy cc-mode */
#endif
}  /* extern "C" { */
#endif

struct cumo_<%=type_name%>_sum_impl {
    __device__ <%=dtype%> Identity(int64_t /*index*/) { return m_zero; }
    __device__ dtype MapIn(dtype in, int64_t /*index*/) { return in; }
    __device__ void Reduce(dtype next, <%=dtype%>& accum) { accum += next; }
    __device__ <%=dtype%> MapOut(<%=dtype%> accum) { return accum; }
};

struct cumo_<%=type_name%>_prod_impl {
    __device__ <%=dtype%> Identity(int64_t /*index*/) { return m_one; }
    __device__ dtype MapIn(dtype in, int64_t /*index*/) { return in; }
    __device__ void Reduce(dtype next, <%=dtype%>& accum) { accum *= next; }
    __device__ <%=dtype%> MapOut(<%=dtype%> accum) { return accum; }
};

struct cumo_<%=type_name%>_min_impl {
    __device__ dtype Identity(int64_t /*index*/) { return DATA_MAX; }
    __device__ dtype MapIn(dtype in, int64_t /*index*/) { return in; }
    __device__ void Reduce(dtype next, dtype& accum) { accum = next < accum ? next : accum; }
    __device__ dtype MapOut(dtype accum) { return accum; }
};

struct cumo_<%=type_name%>_max_impl {
    __device__ dtype Identity(int64_t /*index*/) { return DATA_MIN; }
    __device__ dtype MapIn(dtype in, int64_t /*index*/) { return in; }
    __device__ void Reduce(dtype next, dtype& accum) { accum = next < accum ? accum : next; }
    __device__ dtype MapOut(dtype accum) { return accum; }
};

// TODO(sonots): Implement minmax
__global__ void cumo_<%=type_name%>_ptp_kernel(cumo_na_reduction_arg_t arg)
{
    dtype min=0,max=1;
    //<%=type_name%>_minmax_kernel<<<1,1>>>(n,p1,s1,&min,&max);
    char* p2 = arg.out.ptr;
    *(dtype*)p2 = m_sub(max,min);
}

#if defined(__cplusplus)
extern "C" {
#if 0
} /* satisfy cc-mode */
#endif
#endif

void cumo_<%=type_name%>_sum_kernel_launch(cumo_na_reduction_arg_t* arg)
{
    cumo_reduce<dtype, <%=dtype%>, cumo_<%=type_name%>_sum_impl>(*arg, cumo_<%=type_name%>_sum_impl{});
}

void cumo_<%=type_name%>_prod_kernel_launch(cumo_na_reduction_arg_t* arg)
{
    cumo_reduce<dtype, <%=dtype%>, cumo_<%=type_name%>_prod_impl>(*arg, cumo_<%=type_name%>_prod_impl{});
}

void cumo_<%=type_name%>_min_kernel_launch(cumo_na_reduction_arg_t* arg)
{
    cumo_reduce<dtype, dtype, cumo_<%=type_name%>_min_impl>(*arg, cumo_<%=type_name%>_min_impl{});
}

void cumo_<%=type_name%>_max_kernel_launch(cumo_na_reduction_arg_t* arg)
{
    cumo_reduce<dtype, dtype, cumo_<%=type_name%>_max_impl>(*arg, cumo_<%=type_name%>_max_impl{});
}

void cumo_<%=type_name%>_ptp_kernel_launch(cumo_na_reduction_arg_t* arg)
{
    cumo_<%=type_name%>_ptp_kernel<<<1,1>>>(*arg);
}
