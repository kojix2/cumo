#include "hip/hip_runtime.h"
#if defined(__cplusplus)
#if 0
{ /* satisfy cc-mode */
#endif
}  /* extern "C" { */
#endif

struct cumo_<%=type_name%>_sum_impl {
    __device__ <%=dtype%> Identity(int64_t /*index*/) { return m_zero; }
    __device__ dtype MapIn(dtype in, int64_t /*index*/) { return in; }
    __device__ void Reduce(dtype next, <%=dtype%>& accum) { accum = m_add(next, accum); }
    __device__ <%=dtype%> MapOut(<%=dtype%> accum) { return accum; }
};

struct cumo_<%=type_name%>_prod_impl {
    __device__ <%=dtype%> Identity(int64_t /*index*/) { return m_one; }
    __device__ dtype MapIn(dtype in, int64_t /*index*/) { return in; }
    __device__ void Reduce(dtype next, <%=dtype%>& accum) { accum = m_mul(next, accum); }
    __device__ <%=dtype%> MapOut(<%=dtype%> accum) { return accum; }
};

template<typename Iterator1>
__global__ void cumo_<%=type_name%>_mean_kernel(Iterator1 p1_begin, Iterator1 p1_end, <%=dtype%>* p2, uint64_t n)
{
    dtype init = m_zero;
    dtype sum = thrust::reduce(thrust::cuda::par, p1_begin, p1_end, init, cumo_thrust_plus());
    *p2 = c_div_r(sum, n);
}

template<typename Iterator1>
__global__ void cumo_<%=type_name%>_var_kernel(Iterator1 p1_begin, Iterator1 p1_end, rtype* p2)
{
    cumo_thrust_complex_variance_unary_op<dtype, rtype>  unary_op;
    cumo_thrust_complex_variance_binary_op<dtype, rtype> binary_op;
    cumo_thrust_complex_variance_data<dtype, rtype> init = {};
    cumo_thrust_complex_variance_data<dtype, rtype> result;
    result = thrust::transform_reduce(thrust::cuda::par, p1_begin, p1_end, unary_op, init, binary_op);
    *p2 = result.variance();
}

template<typename Iterator1>
__global__ void cumo_<%=type_name%>_stddev_kernel(Iterator1 p1_begin, Iterator1 p1_end, rtype* p2)
{
    cumo_thrust_complex_variance_unary_op<dtype, rtype>  unary_op;
    cumo_thrust_complex_variance_binary_op<dtype, rtype> binary_op;
    cumo_thrust_complex_variance_data<dtype, rtype> init = {};
    cumo_thrust_complex_variance_data<dtype, rtype> result;
    result = thrust::transform_reduce(thrust::cuda::par, p1_begin, p1_end, unary_op, init, binary_op);
    *p2 = r_sqrt(result.variance());
}

template<typename Iterator1>
__global__ void cumo_<%=type_name%>_rms_kernel(Iterator1 p1_begin, Iterator1 p1_end, rtype* p2, uint64_t n)
{
    rtype init = 0;
    rtype result;
    result = thrust::transform_reduce(thrust::cuda::par, p1_begin, p1_end, cumo_thrust_square(), init, thrust::plus<rtype>());
    *p2 = r_sqrt(result/n);
}

#if defined(__cplusplus)
extern "C" {
#if 0
} /* satisfy cc-mode */
#endif
#endif

void cumo_<%=type_name%>_sum_kernel_launch(cumo_na_reduction_arg_t* arg)
{
    cumo_reduce<dtype, <%=dtype%>, cumo_<%=type_name%>_sum_impl>(*arg, cumo_<%=type_name%>_sum_impl{});
}

void cumo_<%=type_name%>_prod_kernel_launch(cumo_na_reduction_arg_t* arg)
{
    cumo_reduce<dtype, <%=dtype%>, cumo_<%=type_name%>_prod_impl>(*arg, cumo_<%=type_name%>_prod_impl{});
}

void cumo_<%=type_name%>_mean_kernel_launch(uint64_t n, char *p1, ssize_t s1, char *p2)
{
    ssize_t s1_idx = s1 / sizeof(dtype);
    thrust::device_ptr<dtype> data_begin = thrust::device_pointer_cast((dtype*)p1);
    thrust::device_ptr<dtype> data_end   = thrust::device_pointer_cast(((dtype*)p1) + n * s1_idx);
    if (s1_idx == 1) {
        cumo_<%=type_name%>_mean_kernel<<<1,1>>>(data_begin, data_end, (dtype*)p2, n);
    } else {
        cumo_thrust_strided_range<thrust::device_vector<dtype>::iterator> range(data_begin, data_end, s1_idx);
        cumo_<%=type_name%>_mean_kernel<<<1,1>>>(range.begin(), range.end(), (dtype*)p2, n);
    }
}

void cumo_<%=type_name%>_var_kernel_launch(uint64_t n, char *p1, ssize_t s1, char *p2)
{
    ssize_t s1_idx = s1 / sizeof(dtype);
    thrust::device_ptr<dtype> data_begin = thrust::device_pointer_cast((dtype*)p1);
    thrust::device_ptr<dtype> data_end   = thrust::device_pointer_cast(((dtype*)p1) + n * s1_idx);
    if (s1_idx == 1) {
        cumo_<%=type_name%>_var_kernel<<<1,1>>>(data_begin, data_end, (rtype*)p2);
    } else {
        cumo_thrust_strided_range<thrust::device_vector<dtype>::iterator> range(data_begin, data_end, s1_idx);
        cumo_<%=type_name%>_var_kernel<<<1,1>>>(range.begin(), range.end(), (rtype*)p2);
    }
}

void cumo_<%=type_name%>_stddev_kernel_launch(uint64_t n, char *p1, ssize_t s1, char *p2)
{
    ssize_t s1_idx = s1 / sizeof(dtype);
    thrust::device_ptr<dtype> data_begin = thrust::device_pointer_cast((dtype*)p1);
    thrust::device_ptr<dtype> data_end   = thrust::device_pointer_cast(((dtype*)p1) + n * s1_idx);
    if (s1_idx == 1) {
        cumo_<%=type_name%>_stddev_kernel<<<1,1>>>(data_begin, data_end, (rtype*)p2);
    } else {
        cumo_thrust_strided_range<thrust::device_vector<dtype>::iterator> range(data_begin, data_end, s1_idx);
        cumo_<%=type_name%>_stddev_kernel<<<1,1>>>(range.begin(), range.end(), (rtype*)p2);
    }
}

void cumo_<%=type_name%>_rms_kernel_launch(uint64_t n, char *p1, ssize_t s1, char *p2)
{
    ssize_t s1_idx = s1 / sizeof(dtype);
    thrust::device_ptr<dtype> data_begin = thrust::device_pointer_cast((dtype*)p1);
    thrust::device_ptr<dtype> data_end   = thrust::device_pointer_cast(((dtype*)p1) + n * s1_idx);
    if (s1_idx == 1) {
        cumo_<%=type_name%>_rms_kernel<<<1,1>>>(data_begin, data_end, (rtype*)p2, n);
    } else {
        cumo_thrust_strided_range<thrust::device_vector<dtype>::iterator> range(data_begin, data_end, s1_idx);
        cumo_<%=type_name%>_rms_kernel<<<1,1>>>(range.begin(), range.end(), (rtype*)p2, n);
    }
}

