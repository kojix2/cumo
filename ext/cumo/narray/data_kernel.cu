#include "hip/hip_runtime.h"
#include "cumo/narray_kernel.h"

#if defined(__cplusplus)
extern "C" {
#if 0
} /* satisfy cc-mode */
#endif
#endif

__global__ void cumo_iter_copy_bytes_kernel(char *p1, char *p2, ssize_t s1, ssize_t s2, size_t *idx1, size_t *idx2, uint64_t n, ssize_t elmsz)
{
    char *p1_ = NULL;
    char *p2_ = NULL;
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        p1_ = p1 + (idx1 ? idx1[i] : i * s1);
        p2_ = p2 + (idx2 ? idx2[i] : i * s2);
        memcpy(p2_, p1_, elmsz);
    }
}

__global__ void cumo_na_diagonal_index_index_kernel(size_t *idx, size_t *idx0, size_t *idx1, size_t k0, size_t k1, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        idx[i] = idx0[i+k0] + idx1[i+k1];
    }
}

__global__ void cumo_na_diagonal_index_stride_kernel(size_t *idx, size_t *idx0, ssize_t s1, size_t k0, size_t k1, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        idx[i] = idx0[i+k0] + s1*(i+k1);
    }
}

__global__ void cumo_na_diagonal_stride_index_kernel(size_t *idx, ssize_t s0, size_t *idx1, size_t k0, size_t k1, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        idx[i] = s0*(i+k0) + idx1[i+k1];
    }
}

void cumo_iter_copy_bytes_kernel_launch(char *p1, char *p2, ssize_t s1, ssize_t s2, size_t *idx1, size_t *idx2, uint64_t n, ssize_t elmsz)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    cumo_iter_copy_bytes_kernel<<<grid_dim, block_dim>>>(p1, p2, s1, s2, idx1, idx2, n, elmsz);
}

void cumo_na_diagonal_index_index_kernel_launch(size_t *idx, size_t *idx0, size_t *idx1, size_t k0, size_t k1, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    cumo_na_diagonal_index_index_kernel<<<grid_dim, block_dim>>>(idx, idx0, idx1, k0, k1, n);
}

void cumo_na_diagonal_index_stride_kernel_launch(size_t *idx, size_t *idx0, ssize_t s1, size_t k0, size_t k1, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    cumo_na_diagonal_index_stride_kernel<<<grid_dim, block_dim>>>(idx, idx0, s1, k0, k1, n);
}

void cumo_na_diagonal_stride_index_kernel_launch(size_t *idx, ssize_t s0, size_t *idx1, size_t k0, size_t k1, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    cumo_na_diagonal_stride_index_kernel<<<grid_dim, block_dim>>>(idx, s0, idx1, k0, k1, n);
}

#if defined(__cplusplus)
#if 0
{ /* satisfy cc-mode */
#endif
}  /* extern "C" { */
#endif
