#include "hip/hip_runtime.h"
#include "cumo/narray_kernel.h"
#include <stdio.h>

#if defined(__cplusplus)
extern "C" {
#if 0
} /* satisfy cc-mode */
#endif
#endif

__global__ void cumo_iter_copy_bytes_kernel(char *p1, char *p2, ssize_t s1, ssize_t s2, size_t *idx1, size_t *idx2, uint64_t n, ssize_t elmsz)
{
    char *p1_ = NULL;
    char *p2_ = NULL;
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        p1_ = p1 + (idx1 ? idx1[i] : i * s1);
        p2_ = p2 + (idx2 ? idx2[i] : i * s2);
        memcpy(p2_, p1_, elmsz);
    }
}

void cumo_iter_copy_bytes_kernel_launch(char *p1, char *p2, ssize_t s1, ssize_t s2, size_t *idx1, size_t *idx2, uint64_t n, ssize_t elmsz)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    cumo_iter_copy_bytes_kernel<<<grid_dim, block_dim>>>(p1, p2, s1, s2, idx1, idx2, n, elmsz);
}

#if defined(__cplusplus)
#if 0
{ /* satisfy cc-mode */
#endif
}  /* extern "C" { */
#endif
