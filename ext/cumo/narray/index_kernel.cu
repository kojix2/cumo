#include "hip/hip_runtime.h"
#include "cumo/narray_kernel.h"

#if defined(__cplusplus)
extern "C" {
#if 0
} /* satisfy cc-mode */
#endif
#endif

__global__ void cumo_na_index_aref_nadata_index_stride_kernel(size_t *idx, ssize_t s1, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        idx[i] = idx[i] * s1;
    }
}

void cumo_na_index_aref_nadata_index_stride_kernel_launch(size_t *idx, ssize_t s1, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    cumo_na_index_aref_nadata_index_stride_kernel<<<grid_dim, block_dim>>>(idx, s1, n);
}

#if defined(__cplusplus)
#if 0
{ /* satisfy cc-mode */
#endif
}  /* extern "C" { */
#endif

