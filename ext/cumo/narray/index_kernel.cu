#include "hip/hip_runtime.h"
#include "cumo/narray_kernel.h"

#if defined(__cplusplus)
extern "C" {
#if 0
} /* satisfy cc-mode */
#endif
#endif

__global__ void cumo_na_index_aref_nadata_index_stride_kernel(size_t *idx, ssize_t s1, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        idx[i] = idx[i] * s1;
    }
}

__global__ void cumo_na_index_aref_naview_index_index_kernel(size_t *idx, size_t *idx1, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        idx[i] = idx1[idx[i]];
    }
}

__global__ void cumo_na_index_aref_naview_index_stride_last_kernel(size_t *idx, ssize_t s1, size_t last, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        idx[i] = (last - idx[i]) * s1;
    }
}

__global__ void cumo_na_index_aref_naview_index_stride_kernel(size_t *idx, ssize_t s1, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        idx[i] = idx[i] * s1;
    }
}

__global__ void cumo_na_index_aref_naview_index_index_beg_step_kernel(size_t *idx, size_t *idx1, size_t beg, ssize_t step, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        idx[i] = idx1[beg + step * i];
    }
}

void cumo_na_index_aref_nadata_index_stride_kernel_launch(size_t *idx, ssize_t s1, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    cumo_na_index_aref_nadata_index_stride_kernel<<<grid_dim, block_dim>>>(idx, s1, n);
}

void cumo_na_index_aref_naview_index_index_kernel_launch(size_t *idx, size_t *idx1, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    cumo_na_index_aref_naview_index_index_kernel<<<grid_dim, block_dim>>>(idx, idx1, n);
}

void cumo_na_index_aref_naview_index_stride_last_kernel_launch(size_t *idx, ssize_t s1, size_t last, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    cumo_na_index_aref_naview_index_stride_last_kernel<<<grid_dim, block_dim>>>(idx, s1, last, n);
}

void cumo_na_index_aref_naview_index_stride_kernel_launch(size_t *idx, ssize_t s1, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    cumo_na_index_aref_naview_index_stride_kernel<<<grid_dim, block_dim>>>(idx, s1, n);
}

void cumo_na_index_aref_naview_index_index_beg_step_kernel_launch(size_t *idx, size_t *idx1, size_t beg, ssize_t step, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    cumo_na_index_aref_naview_index_index_beg_step_kernel<<<grid_dim, block_dim>>>(idx, idx1, beg, step, n);
}

#if defined(__cplusplus)
#if 0
{ /* satisfy cc-mode */
#endif
}  /* extern "C" { */
#endif

